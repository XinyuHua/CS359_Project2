#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

__global__ void SetMatrixA( float *d_a )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	d_a[idx] = blockIdx.x + 0.1 * threadIdx.x;
}

__global__ void SetMatrixB( float *d_a )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	d_a[idx] = 3 * blockIdx.x + 0.5 * threadIdx.x;
}

// Warning: While use this function, dim should not exceed 32
__global__ void MultiplicationSingle( float *d_a, float *d_b, float *d_c, int dim)
{
	int idx = threadIdx.x;
	int row = threadIdx.x / dim;
	int col = threadIdx.x % dim;
	float rst = 0;
	for(int i = 0; i < dim; ++i)
	{
		rst = d_a[row * dim + i]*d_b[i * dim + col] + rst;
	}
	d_c[row * dim + col] = rst;
}

__global__ void MultiplicationMulti( float *d_a, float *d_b, float *d_c)
{
	int dim = blockDim.x;
	int row = blockIdx.x;
	int col = threadIdx.x;
	float rst = 0;
	for(int i = 0; i < dim; ++i)
	{
	    rst = d_a[row * dim + i]*d_b[i * dim + col] + rst;
	}
	d_c[row * dim + col] = rst;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

typedef struct{
	int width;
	int height;
	float *vals;
}Matrix;

void displayMatrix(Matrix m)
{
	printf("----------------------------------\n");
	for(int i = 0; i < m.height; ++i)
	{	
		printf("Row %d\n",i);
		for(int j = 0; j < m.width; ++j)
		{
			printf("%.2f\n", m.vals[m.width * i + j]);
		}
		printf("\n\n");
	}
}


int main( int argc, char** argv) 
{
	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;
	
	int dim = 64;
	
	int numBlocks = dim;
	int numThreadsPerBlock = dim;
	
	size_t memSize = numBlocks * numThreadsPerBlock * sizeof(float);
	
	h_a = (float *) malloc(memSize);
	h_b = (float *) malloc(memSize);
	h_c = (float *) malloc(memSize);

    hipMalloc( &d_a, memSize );
	hipMalloc( &d_b, memSize );
	hipMalloc( &d_c, memSize );

//  Initialize Matrix a and Matrix b
	dim3 dimGrid( numBlocks );
    dim3 dimBlock( numThreadsPerBlock  );
    
	SetMatrixA<<< numBlocks , numThreadsPerBlock >>>( d_a );
	hipDeviceSynchronize();

	SetMatrixB<<< numBlocks , numThreadsPerBlock >>>( d_b );
	hipDeviceSynchronize();

	checkCUDAError("kernel execution");
	hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost  );
	hipMemcpy(h_b, d_b, memSize, hipMemcpyDeviceToHost  );
	checkCUDAError("hipMemcpy");

//  Perform the Multiplication
	MultiplicationMulti<<< dim, dim >>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	checkCUDAError("kernel execution");
	hipMemcpy(h_c, d_c, memSize, hipMemcpyDeviceToHost  );
	checkCUDAError("hipMemcpy");

	Matrix ma = {dim,dim,h_a};
	Matrix mb = {dim,dim,h_b};
	Matrix mc = {dim,dim,h_c};
	//displayMatrix(ma);
	//displayMatrix(mb);
	displayMatrix(mc);
	
	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
    hipDeviceReset();
	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
